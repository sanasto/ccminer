#include <hip/hip_runtime.h>



#include <stdio.h>
#include <memory.h>

#define W_ALIGNMENT 65

// Folgende Definitionen sp�ter durch header ersetzen
typedef unsigned int uint32_t;

// globaler Speicher f�r alle HeftyHashes aller Threads
extern uint32_t *d_heftyHashes[8];
extern uint32_t *d_nonceVector[8];

// globaler Speicher f�r unsere Ergebnisse
uint32_t *d_hash2output[8];


/* Hash-Tabellen */
__constant__ uint32_t sha256_gpu_constantTable[64];

// muss expandiert werden
__constant__ uint32_t sha256_gpu_blockHeader[16]; // 2x512 Bit Message
__constant__ uint32_t sha256_gpu_register[8];

uint32_t sha256_cpu_hashTable[] = { 0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19 };
uint32_t sha256_cpu_constantTable[] = {
	0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
	0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
	0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
	0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
	0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
	0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
	0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
	0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2,
};

#define S(x, n)			(((x) >> (n)) | ((x) << (32 - (n))))
#define R(x, n)			((x) >> (n))
#define Ch(x, y, z)		((x & (y ^ z)) ^ z)
#define Maj(x, y, z)	((x & (y | z)) | (y & z))
#define S0(x)			(S(x, 2) ^ S(x, 13) ^ S(x, 22))
#define S1(x)			(S(x, 6) ^ S(x, 11) ^ S(x, 25))
#define s0(x)			(S(x, 7) ^ S(x, 18) ^ R(x, 3))
#define s1(x)			(S(x, 17) ^ S(x, 19) ^ R(x, 10))

#define SWAB32(x)		( ((x & 0x000000FF) << 24) | ((x & 0x0000FF00) << 8) | ((x & 0x00FF0000) >> 8) | ((x & 0xFF000000) >> 24) )

// Die Hash-Funktion
__global__ void sha256_gpu_hash(int threads, uint32_t startNounce, void *outputHash, uint32_t *heftyHashes, uint32_t *nonceVector)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Z�hler
		uint32_t nounce = startNounce + thread;
		nonceVector[thread] = nounce;
	
		// jeder thread in diesem  Block bekommt sein eigenes W Array im Shared memory
		//extern __shared__ unsigned char s[];
		//uint32_t *W = (uint32_t *)(&s[W_ALIGNMENT * sizeof(uint32_t) * threadIdx.x]);
		uint32_t W1[16];
		uint32_t W2[16];

		// Initialisiere die register a bis h mit der Hash-Tabelle
		uint32_t regs[8];
		uint32_t hash[8];

		// pre
#pragma unroll 8
		for (int k=0; k < 8; k++)
		{
			regs[k] = sha256_gpu_register[k];
			hash[k] = regs[k];
		}
	
		// 2. Runde
		//memcpy(W, &sha256_gpu_blockHeader[0], sizeof(uint32_t) * 16); // TODO: aufsplitten in zwei Teilbl�cke
		//memcpy(&W[5], &heftyHashes[8 * (blockDim.x * blockIdx.x + threadIdx.x)], sizeof(uint32_t) * 8); // den richtigen Hefty1 Hash holen		
#pragma unroll 16
		for(int k=0;k<16;k++)
			W1[k] = sha256_gpu_blockHeader[k];

		uint32_t offset = 8 * (blockDim.x * blockIdx.x + threadIdx.x);
#pragma unroll 8
		for(int k=0;k<8;k++)
			W1[5+k] = heftyHashes[offset + k];


#pragma unroll 8
		for (int i=5; i <5+8; ++i) W1[i] = SWAB32(W1[i]); // die Hefty1 Hashes brauchen eine Drehung ;)
		W1[3] = SWAB32(nounce);

// Progress W1
#pragma unroll 16
		for(int j=0;j<16;j++)
		{
			uint32_t T1, T2;
			T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_gpu_constantTable[j] + W1[j];
			T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
			#pragma unroll 7
			for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
			regs[0] = T1 + T2;
			regs[4] += T1;
		}

// Progress W2...W3
#pragma unroll 3
		for(int k=0;k<3;k++)
		{
	#pragma unroll 2
			for(int j=0;j<2;j++)
				W2[j] = s1(W1[14+j]) + W1[9+j] + s0(W1[1+j]) + W1[j];
	#pragma unroll 5
			for(int j=2;j<7;j++)
				W2[j] = s1(W2[j-2]) + W1[9+j] + s0(W1[1+j]) + W1[j];

	#pragma unroll 8
			for(int j=7;j<15;j++)
				W2[j] = s1(W2[j-2]) + W2[j-7] + s0(W1[1+j]) + W1[j];

			W2[15] = s1(W2[13]) + W2[8] + s0(W2[0]) + W1[15];

			// Rundenfunktion
	#pragma unroll 16
			for(int j=0;j<16;j++)
			{
				uint32_t T1, T2;
				T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_gpu_constantTable[j + 16 * (k+1)] + W2[j];
				T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
				#pragma unroll 7
				for (int l=6; l >= 0; l--) regs[l+1] = regs[l];
				regs[0] = T1 + T2;
				regs[4] += T1;
			}

	#pragma unroll 16
			for(int j=0;j<16;j++)
				W1[j] = W2[j];
		}

/*
		for(int j=16;j<64;j++)
			W[j] = s1(W[j-2]) + W[j-7] + s0(W[j-15]) + W[j-16];
	
#pragma unroll 64
		for(int j=0;j<64;j++)
		{
			uint32_t T1, T2;
			T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_gpu_constantTable[j] + W[j];
			T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
			#pragma unroll 7
			for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
			regs[0] = T1 + T2;
			regs[4] += T1;
		}
*/
#pragma unroll 8
		for(int k=0;k<8;k++)
			hash[k] += regs[k];

#pragma unroll 8
		for(int k=0;k<8;k++)
			((uint32_t*)outputHash)[8*thread+k] = SWAB32(hash[k]);
	}
}

// Setup-Funktionen
__host__ void sha256_cpu_init(int thr_id, int threads)
{
	// Kopiere die Hash-Tabellen in den GPU-Speicher
	hipMemcpyToSymbol(	HIP_SYMBOL(sha256_gpu_constantTable),
						sha256_cpu_constantTable,
						sizeof(uint32_t) * 64 );

	// Speicher f�r alle Ergebnisse belegen
	hipMalloc(&d_hash2output[thr_id], 8 * sizeof(uint32_t) * threads);
}

__host__ void sha256_cpu_setBlock(void *data)
	// data muss 84-Byte haben!
	// heftyHash hat 32-Byte
{
	// Nachricht expandieren und setzen
	uint32_t msgBlock[32];

	memset(msgBlock, 0, sizeof(uint32_t) * 32);
	memcpy(&msgBlock[0], data, 84);
	memset(&msgBlock[21], 0, 32); // vorl�ufig  Nullen anstatt der Hefty1 Hashes einf�llen
	msgBlock[29] |= 0x80;
	msgBlock[31] = 928; // bitlen
	
	for(int i=0;i<31;i++) // Byteorder drehen
		msgBlock[i] = SWAB32(msgBlock[i]);

	// die erste Runde wird auf der CPU durchgef�hrt, da diese f�r
	// alle Threads gleich ist. Der Hash wird dann an die Threads
	// �bergeben
	uint32_t W[64];

	// Erstelle expandierten Block W
	memcpy(W, &msgBlock[0], sizeof(uint32_t) * 16);	
	for(int j=16;j<64;j++)
		W[j] = s1(W[j-2]) + W[j-7] + s0(W[j-15]) + W[j-16];

	// Initialisiere die register a bis h mit der Hash-Tabelle
	uint32_t regs[8];
	uint32_t hash[8];

	// pre
    for (int k=0; k < 8; k++)
	{
		regs[k] = sha256_cpu_hashTable[k];
		hash[k] = regs[k];
	}

	// 1. Runde
	for(int j=0;j<64;j++)
	{
		uint32_t T1, T2;
		T1 = regs[7] + S1(regs[4]) + Ch(regs[4], regs[5], regs[6]) + sha256_cpu_constantTable[j] + W[j];
		T2 = S0(regs[0]) + Maj(regs[0], regs[1], regs[2]);
		
		//#pragma unroll 7
		for (int k=6; k >= 0; k--) regs[k+1] = regs[k];
		// sollte mal noch durch memmov ersetzt werden!
//		memcpy(&regs[1], &regs[0], sizeof(uint32_t) * 7);
		regs[0] = T1 + T2;
		regs[4] += T1;
	}

	for(int k=0;k<8;k++)
		hash[k] += regs[k];

	// hash speichern
	hipMemcpyToSymbol(	HIP_SYMBOL(sha256_gpu_register),
						hash,
						sizeof(uint32_t) * 8 );

	// Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
	hipMemcpyToSymbol(	HIP_SYMBOL(sha256_gpu_blockHeader),
						&msgBlock[16],
						64);
}

__host__ void sha256_cpu_copyHeftyHash(int thr_id, int threads, void *heftyHashes, int copy)
{
	// Hefty1 Hashes kopieren
	if (copy) hipMemcpy( d_heftyHashes[thr_id], heftyHashes, 8 * sizeof(uint32_t) * threads, hipMemcpyHostToDevice );
	//else cudaThreadSynchronize();
}

__host__ void sha256_cpu_hash(int thr_id, int threads, int startNounce)
{
	const int threadsperblock = 128;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs (abh�ngig von der Threadanzahl)
	//size_t shared_size = W_ALIGNMENT*sizeof(uint32_t)*threadsperblock;  // ein uint32_t eingef�gt gegen Bank Konflikte
	size_t shared_size = 0;

//	fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);

	sha256_gpu_hash<<<grid, block, shared_size>>>(threads, startNounce, d_hash2output[thr_id], d_heftyHashes[thr_id], d_nonceVector[thr_id]);
}
